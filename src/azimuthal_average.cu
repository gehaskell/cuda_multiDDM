// Code draws on Nvidia SDK reduction_example

#include <string>
#include <iostream>
#include <fstream>

#include "constants.hpp"
#include "debug.hpp"
#include "reduction.h"


inline unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

///////////////////////////////////////////////////////
//	Writes I(q, tau) to file. The format of the
//	output is described in detail in the documentation.
///////////////////////////////////////////////////////
void writeIqtToFile(std::string filename,
					float *iqtau,
					float *q_arr, int q_count,
					int *tau_arr, int tau_count,
					int fps) {

    std::ofstream out_file(filename); // attempt to open file

    if (out_file.is_open()) {
    	// q - values
    	for (int qi = 0; qi < q_count; qi++) {
    		out_file << q_arr[qi] << " ";
    	}

    	out_file << "\n";

    	// tau - values
    	for (int ti = 0; ti < tau_count; ti++) {
    		out_file << static_cast<float>(tau_arr[ti]) / static_cast<float>(fps) << " ";
    	}

    	out_file << "\n";

    	// I(q, tau) - values
		for (int qi = 0; qi < q_count; qi++) {
	    	for (int ti = 0; ti < tau_count; ti++) {
	    		out_file << iqtau[qi * tau_count + ti] << " ";
	    	}
	    	out_file << "\n";
		}

		out_file.close();
		verbose("I(Q, tau) written to %s\n", filename.c_str());
    } else {
		fprintf(stderr, "[Out Error] Unable to open %s.\n", filename.c_str());
		exit(EXIT_FAILURE);
    }
}

///////////////////////////////////////////////////////
//	This function performs azimuthal averaging on the
//	host (i.e. CPU), in almost all cases this is far slower
//	than using the GPU, included for completeness.
///////////////////////////////////////////////////////
void analyseFFTHost(std::string filename,
					float *d_data_in,
					float norm_factor,
					float *q_arr, int q_count,
					int *tau_arr, int tau_count,
					float q_tolerance,
					int w, int h,
					int tile_index,
					int fps) {

	float * iqtau = new float[tau_count * q_count];

	float q2_arr[q_count]; // array containing squared q-values
	for (int i = 0; i < q_count; i++)
		q2_arr[i] = q_arr[i] * q_arr[i];

	int element_count = (w/2 + 1) * h; // number of elements in mask

	// pre-calc some values
	float tol2 = q_tolerance * q_tolerance;
	int half_w = w / 2;
	int half_h = h / 2;

	int x_shift, y_shift;
	float r2, r2q2_ratio;

    for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
		for (int q_idx = 0; q_idx < q_count; q_idx++) {
			float val = 0;
			float px_count = 0;

			for (int x = 0; x < (w/2 + 1); x++) {
				for (int y = 0; y < h; y++) {
					// manual FFT shift
					x_shift = (x + half_w) % w;
					y_shift = (y + half_h) % h;

					// distance from centre
					x_shift -= half_w;
					y_shift -= half_h;

					r2 = x_shift * x_shift + y_shift * y_shift;
					r2q2_ratio = r2 / q2_arr[q_idx];

					// element true if r in range [1.0 q, q_tolerance * q]
					if (1 <= r2q2_ratio && r2q2_ratio <= tol2) {
						val += d_data_in[element_count * tau_idx * tile_index + y * (w/2 + 1) + x];
						px_count += 1.0;
					}

		    		val *= 2; // account for symmetry
		            val /= px_count;
		            val /= norm_factor;

					iqtau[q_idx * tau_count + tau_idx] = val;
				}
			}
		}
    }

    // Finally write I(q, tau) to file
    writeIqtToFile(filename, iqtau, q_arr, q_count, tau_arr, tau_count, fps);
}

///////////////////////////////////////////////////////
//	This function builds azimuthal boolean pixel masks
//	based on given input parameters. Masks are built on
//	host and copied to given device memory location.
///////////////////////////////////////////////////////
void buildAzimuthMask(bool *d_mask_out,
					  int *h_pixel_counts,
					  float *q_arr, int q_count,
					  float q_tolerance,
					  int w, int h) {

	float q2_arr[q_count]; // array containing squared q-values
	for (int i = 0; i < q_count; i++)
		q2_arr[i] = q_arr[i] * q_arr[i];

	int element_count = (w/2 + 1) * h; // number of elements in mask
	bool *h_mask = new bool[element_count * q_count];

	// pre-calc some values
	float tol2 = q_tolerance * q_tolerance;
	int half_w = w / 2;
	int half_h = h / 2;

	int x_shift, y_shift;
	float r2, r2q2_ratio;

	bool px;
	for (int q_idx = 0; q_idx < q_count; q_idx++) {
		h_pixel_counts[q_idx] = 0;

		for (int x = 0; x < (w/2 + 1); x++) {
			for (int y = 0; y < h; y++) {
				// manual FFT shift
				x_shift = (x + half_w) % w;
				y_shift = (y + half_h) % h;

				// distance from centre
				x_shift -= half_w;
				y_shift -= half_h;

				r2 = x_shift * x_shift + y_shift * y_shift;
				r2q2_ratio = r2 / q2_arr[q_idx];

				// element true if r in range [1.0 q, q_tolerance * q]
				px = (1 <= r2q2_ratio) && (r2q2_ratio <= tol2);
                if (px) h_pixel_counts[q_idx] += 1;
                h_mask[q_idx * element_count + y * (w/2 + 1) + x] = px;
			}
		}

		if (h_pixel_counts[q_idx] == 0) {
			verbose("[Mask Generation] q: %f, (#q: %d) has zero mask pixels for scale %d x %d\n", q_arr[q_idx], q_idx, w, h);
		}

    }
	// Copy mask onto GPU
    gpuErrorCheck(hipMemcpy(d_mask_out, h_mask, sizeof(bool) * element_count * q_count, hipMemcpyHostToDevice));
    delete h_mask;

}

///////// NVIDIA ////////

// For now just use whole mask, in future could investigate
// Performing 2 reductions on (w / 2) * (h / 2) as this would
// most likely be a power of 2, for which reduction most optimised
void analyseFFTDevice(std::string filename,
					  float *d_data_in,
					  bool *d_mask,
					  int *h_px_count,
					  float norm_factor,
					  int *tau_arr, int tau_count,
					  float *q_label_arr, int q_count,
					  int tile_count,
					  int width,
					  int height,
					  int fps) {

	// TODO: move to boarder function
	// get device capability, to avoid block/grid size exceed the upper bound
	hipDeviceProp_t prop;
	int device;
	gpuErrorCheck(hipGetDevice(&device));
	gpuErrorCheck(hipGetDeviceProperties(&prop, device));

	int n = (width / 2 + 1) * height;

	// Compute the number of threads and blocks to use for the given reduction
	// kernel For the kernels >= 3, we set threads / block to the minimum of
	// maxThreads and n/2.

	int threads = (n < BLOCKSIZE * 2) ? nextPow2((n + 1) / 2) : BLOCKSIZE;
	int blocks = (n + (threads * 2 - 1)) / (threads * 2);

	if ((float)threads * blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock) {
		printf("[Reduction] Image is too large.\n");
		exit(EXIT_FAILURE);
	}

	if (blocks > prop.maxGridSize[0]) {
		printf("[Reduction] Grid size <%d> exceeds the device capability <%d>, set block size as "
				"%d (original %d)\n", blocks, prop.maxGridSize[0], threads * 2, threads);

		blocks /= 2;
		threads *= 2;
	}

	blocks = (64 < blocks) ? 64 : blocks;

	float *d_intermediateSums;
	float *h_intermediateSums = new float[blocks];

	gpuErrorCheck(hipMalloc((void **)&d_intermediateSums, sizeof(float) * blocks));


	float * iq_tau = new float[tau_count * q_count]();
	for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
		for (int q_idx = 0; q_idx < q_count; q_idx++) {

			float val = 0;

			if (h_px_count[q_idx] != 0) {

				// execute the kernel
				maskReduce<float>(n, threads, blocks, d_data_in + n*tau_idx*tile_count, d_mask + n*q_idx, d_intermediateSums);

				// check if kernel execution generated an error
				//getLastCudaError("Kernel execution failed");

				// sum partial sums from each block on CPU TODO can do this on device too
				// copy result from device to host
				gpuErrorCheck(hipMemcpy(h_intermediateSums, d_intermediateSums, blocks * sizeof(float),
							 hipMemcpyDeviceToHost));

				for (int i = 0; i < blocks; i++) {
					val += h_intermediateSums[i];
				}

				val *= 2; // account for symmetry
				val /= static_cast<float>(h_px_count[q_idx]);
				val /= norm_factor;
			}

        	iq_tau[q_idx * tau_count + tau_idx] = val;
		}
	}

	hipDeviceSynchronize();

    // Finally write I(q, tau) to file
    writeIqtToFile(filename, iq_tau, q_label_arr, q_count, tau_arr, tau_count, fps);


}

///////// NVIDIA END ////////


